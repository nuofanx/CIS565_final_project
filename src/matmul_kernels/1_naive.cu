#include "hip/hip_runtime.h"
#pragma once

#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>


// A is input matrix, B is weight matrix 
// matmul operation on matrix A of shape (M, K) and matrix B of shape (N, K)
// resulting matrix C is of shape (M, N)
__global__ void matmulKernel_naive(void* C, void* A, void* B, int M, int N, int K, int weight_quant_num){
    // use strided in-memory representation index x and y 
    const uint x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint y = blockIdx.y * blockDim.y + threadIdx.y;
    // check if current thread is used in the calculation 
    if (x< M && y < N){
        // init tmp val 
        float tmp = 0.0f;
        // loop through each col of A and row of B, with K elements  
        for (int i = 0; i < K; i++) {
            // x is row index of A and C and y is col index of B and C
            // type cast to float as input might be half precision  
            tmp += (float)A[x * K + i] * (float)B[i* N + y];
        }
        // store the result to C
        
        switch (weight_quant_num){
            case 0:
                C[x*K + y] = tmp;
                break;
            case 1:
                C[x*K + y] = (half) tmp;
                break;
            default:
                throw std::invalid_argument("Unknown weight quantization number");
        }
    }
}


