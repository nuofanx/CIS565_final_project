#include "hip/hip_runtime.h"
#pragma once

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// Vectorize SMEM and GMEM Accesses
__global__ void matmulKernel_vecSMEM_GMEM(){
    float4 tmp =
        reinterpret_cast<float4 *>(&A[innerRowA * K + innerColA * 4])[0];
    // transpose A during the GMEM to SMEM transfer
    As[(innerColA * 4 + 0) * BM + innerRowA] = tmp.x;
    As[(innerColA * 4 + 1) * BM + innerRowA] = tmp.y;
    As[(innerColA * 4 + 2) * BM + innerRowA] = tmp.z;
    As[(innerColA * 4 + 3) * BM + innerRowA] = tmp.w;

    //why faster than just manually unrolling the access (or using pragma unroll)
    reinterpret_cast<float4 *>(&Bs[innerRowB * BN + innerColB * 4])[0] =
        reinterpret_cast<float4 *>(&B[innerRowB * N + innerColB * 4])[0];
    __syncthreads();

    // the compiler has no way to verify that the float* B pointer that is passed to the kernel is 128b aligned, which would be a requirement for using LDG.E.128. So the reinterpret_cast’s only purpose is to promise the compiler that the float* B pointer will be aligned.
    // Bs[innerRowB * BN + innerColB * 4 + 0] = B[innerRowB * N + innerColB * 4 + 0];
    // Bs[innerRowB * BN + innerColB * 4 + 1] = B[innerRowB * N + innerColB * 4 + 1];
    // Bs[innerRowB * BN + innerColB * 4 + 2] = B[innerRowB * N + innerColB * 4 + 2];
    // Bs[innerRowB * BN + innerColB * 4 + 3] = B[innerRowB * N + innerColB * 4 + 3];
}